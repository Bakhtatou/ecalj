#include "hip/hip_runtime.h"
HOST_INFORMATION platform: gfortran
HOST_INFORMATION compiler version: gcc バージョン 4.6.1 (Ubuntu/Linaro 4.6.1-9ubuntu3) 
HOST_INFORMATION FFLAGS (<=120): -O2 -fomit-frame-pointer -funroll-loops -ffast-math -ffixed-line-length-132 -DHASIARGC -DHASGETARG -DFDATE -DHASCPUTIME 
HOST_INFORMATION LIBLOC (<=120): /usr/lib/libfftw3.so.3 /usr/lib/liblapack.so.3gf /usr/lib/libblas.so.3gf
HOST_INFORMATION uname -a (<=120): Linux TT4 3.0.0-20-generic #34-Ubuntu SMP Tue May 1 17:24:39 UTC 2012 x86_64 x86_64 x86_64 GNU/Linux
HOST_INFORMATION /etc/issue: Ubuntu 11.10 \n \l
HOST_INFORMATION git branch: refs/heads/newaniso
HOST_INFORMATION git commit: ff4f0b57505a84e0eaabe44d640190810c88782c
HOST_INFORMATION linked at: Tue Jun 19 14:13:38 JST 2012
 -----------------------  START LMF      -----------------------
 ptenv() is called with EXT=cu
 ptenv() not supported, but continue.
 HEADER Cu
 Cu       xxx            1           1

 rdctrl: reset global max nl from 5 to 6
  mxcst switch =           1           0 F F F
  LMF  vn 7.00(LMF 7.0)  verb 31,20
 special:  APW basis
 bz:       metal(3), tetra, invit, fixed-spin-mom 
 goto setcg
 lattic:

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-08   alat= 6.79800   awald= 0.467
         r1=  1.959   nkd= 135      q1=  5.910   nkg= 181

 SGROUP: 1 symmetry operations from 0 generators
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 GROUPG: the following are sufficient to generate the space group:
         i*r3(1,1,-1) r4x
         i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 zzz nclass=           1
 
 lstar xxx=          -2
 BZMESH:  29 irreducible QP from 512 ( 8 8 8 )  shift= F F F
 lstar xxx=          -2
 TETIRR: sorting 3072 tetrahedra ...
 76 inequivalent ones found

 species data:  augmentation                           density
 spec       rmt   rsma lmxa kmxa      lmxl     rg   rsmv  kmxv foca   rfoca
 Cu       2.280  0.912    5    5         5  0.570  1.140    15    1   0.912

 gvlist: cutoff radius   9.803 gives   1243   recips of max   3375
 SGVSYM: 53 symmetry stars found for 1243 reciprocal lattice vectors
 

 Makidx:  hamiltonian dimensions Low, Int, High, Negl: 9 0 27 0
 suham :  36 augmentation channels, 36 local potential channels  Maximum lmxa=5

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  Cu       0    1.50  -0.28   4.956    2.32E-06     169 
  Cu       1    1.50  -0.10   5.245    1.10E-06     181 
  Cu       2    0.95  -0.11   8.973    1.81E-06     941 

 suham:  q-dependent PW basis with  Emin = 0 < E < 5.
         Est. min,max PW dimension = 12,18.  Use npwpad = 3 => ndham = 30

 iors  : read restart file (binary, mesh density) 
 iors  : empty file ... nothing read

 rdovfa: read and overlap free-atom densities (mesh density) ...
 rdovfa: expected Cu,      read Cu       with rmt=  2.2800  mesh   655  0.015

 ovlpfa: overlap smooth part of FA densities
 site   1  spec  1  pos  0.0000  0.0000  0.0000  Qsmooth 4.728719
 total smooth Q = 4.728719

 Free atom and overlapped crystal site charges:
   ib    true(FA)    smooth(FA)  true(OV)    smooth(OV)    local
    1    9.758197    3.486916   10.209422    3.938142    6.271281

 Smooth charge on mesh:            4.728719
 Sum of local charges:             6.271281
 Total valence charge:            11.000000
 Sum of core charges:             18.000000
 Sum of nuclear charges:         -29.000000
 Homogeneous background:           0.000000
 Deviation from neutrality:        0.000000

 --- BNDFP:  begin iteration 1 of 20 ---
 ttt nevmx w=           0  5.00000000000000010E-003

 avg es pot at rmt= 0.554288  avg sphere pot= 0.625098  vconst=-0.554288
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           1           5 T

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===
 potential shift to crystal energy zero:    0.000086


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 end of suham2
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F

  --- Hamiltonian index ---
  ib l  k offl(iorb)+1  offl(iorb)+2*l+1  trim(spec)
  ngrp=     -999999
  end of hambls mode=           0
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.95D-06    2  0.57D-03    3  0.57D-03    4  0.57D-03    5  0.40D-01
    6  0.40D-01    7  0.51D-01    8  0.90D-01    9  0.90D-01   10  0.90D-01
   11  0.40D+00   12  0.40D+00   13  0.40D+00   14  0.47D+00   15  0.59D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.6650 -0.0472 -0.0472 -0.0472  0.0270  0.0270  1.7100  1.9162  1.9162
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4185 -0.0983 -0.0478 -0.0233  0.0207  0.0764  0.8953  1.2978  1.7913
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.2385 -0.1775  0.0760  0.0911  0.0911  0.1426  0.6393  1.0060  1.0060

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.151737;  11.000000 electrons
         Sum occ. bands:   -0.756612, incl. Bloechl correction: -0.006665

 Saved qp weights ...
 Start second band pass ...
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.95D-06    2  0.57D-03    3  0.57D-03    4  0.57D-03    5  0.40D-01
    6  0.40D-01    7  0.51D-01    8  0.90D-01    9  0.90D-01   10  0.90D-01
   11  0.40D+00   12  0.40D+00   13  0.40D+00   14  0.47D+00   15  0.59D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.6650 -0.0472 -0.0472 -0.0472  0.0270  0.0270  1.7100  1.9162  1.9162
 (warning) DOS window (-1,0) reset to (-1.1650,0.6517)
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4185 -0.0983 -0.0478 -0.0233  0.0207  0.0764  0.8953  1.2978  1.7913
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.2385 -0.1775  0.0760  0.0911  0.0911  0.1426  0.6393  1.0060  1.0060

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.151737;  11.000000 electrons
         Sum occ. bands:   -0.756612, incl. Bloechl correction: -0.006665

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1    9.841195    3.278269    6.562926

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.488975   -0.340547    4.690000    4.656558    4.500000    4.656558
 1     0    0.510456   -0.159501    4.420000    4.397639    4.250000    4.397639
 2     0    8.810559   -0.031904    3.880000    3.867564    3.147584    3.867564
 3     0    0.024903   -0.052670    4.120000    4.125488    4.102416    4.125488
 4     0    0.005037   -0.038215    5.100000    5.087103    5.077979    5.087103
 5     0    0.001265   -0.016834    6.100000    6.067507    6.062833    6.067507

 Harris energy:
 sumev=       -0.756612  val*vef=    -189.393534   sumtv=     188.636922
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756520
 rhoeps=    -130.258096     utot=   -6534.971163    ehar=   -3304.835818

 avg es pot at rmt= 0.689901  avg sphere pot= 0.633467  vconst=-0.689901
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           0           5 F

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=F
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===


 ekin=3344.586478  rho*v=-6649.079153 ehf =-3304.835818  ehks =-3304.492675
 mixrho: sum smrho  init = 0.203205D+03-0.693212D-28 0.203205D+03       0
 mixrho: sum smrnew new  = 0.190672D+03 0.244466D-17 0.190672D+03       0
  
 mixrho: dqsum rmsuns= -0.37134D-02  0.45470D-01  0.13184D-18
 mixrealsmooth= T
 wgtsmooth=  1.72132593164774084E-002
 mixrho:  sought 8 iter from file mixm; read 0.  RMS DQ=3.44e-2
 charges:       old           new         screened      rms diff       lin mix
 smooth       4.728719      4.437074      4.437074      0.045470      4.437074
 site    1    6.271281      6.562926      6.562926      0.014872      6.562926
 AMIX: nmix=0 mmix=8  nelts=5333  beta=1  tm=5  rmsdel=3.44e-2
 mixrealsmooth= T
 smrho qcell: add correction to smrho= -5.57011645696547930E-008 -7.09219692408622063E-010
 mixrho: all smrho is positive for isp=           1

 iors  : write restart file (binary, mesh density) 

   it  1  of 20    ehf=   -3304.835818   ehk=   -3304.492675
h ehf=-3304.8358175 ehk=-3304.4926747

 --- BNDFP:  begin iteration 2 of 20 ---
 ttt nevmx w=           0  5.00000000000000010E-003

 avg es pot at rmt= 0.689901  avg sphere pot= 0.633467  vconst=-0.689901
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           1           5 T

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===
 potential shift to crystal energy zero:    0.000110


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 end of suham2
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.93D-06    2  0.33D-03    3  0.33D-03    4  0.33D-03    5  0.21D-01
    6  0.21D-01    7  0.31D-01    8  0.45D-01    9  0.45D-01   10  0.45D-01
   11  0.24D+00   12  0.24D+00   13  0.24D+00   14  0.27D+00   15  0.35D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.9319 -0.9319 -0.9319 -0.8970 -0.8970 -0.8327  1.6065  1.7159  1.7159
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.9461 -0.9274 -0.9181 -0.8984 -0.8907 -0.5508  0.6217  1.0694  1.6576
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.9790 -0.9711 -0.8735 -0.8689 -0.8689 -0.0335  0.2689  0.8090  0.8090

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.252804;  11.000000 electrons
         Sum occ. bands:   -9.639146, incl. Bloechl correction: -0.013835

 Saved qp weights ...
 Start second band pass ...
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.93D-06    2  0.33D-03    3  0.33D-03    4  0.33D-03    5  0.21D-01
    6  0.21D-01    7  0.31D-01    8  0.45D-01    9  0.45D-01   10  0.45D-01
   11  0.24D+00   12  0.24D+00   13  0.24D+00   14  0.27D+00   15  0.35D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.9319 -0.9319 -0.9319 -0.8970 -0.8970 -0.8327  1.6065  1.7159  1.7159
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.9461 -0.9274 -0.9181 -0.8984 -0.8907 -0.5508  0.6217  1.0694  1.6576
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.9790 -0.9711 -0.8735 -0.8689 -0.8689 -0.0335  0.2689  0.8090  0.8090

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.252804;  11.000000 electrons
         Sum occ. bands:   -9.639146, incl. Bloechl correction: -0.013835

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.453936    1.990699    8.463236

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.421972   -0.548056    4.656558    4.675732    4.500000    4.675732
 1     0    0.220875   -0.490003    4.397639    4.366703    4.250000    4.366703
 2     0    9.803395   -0.912551    3.867564    3.908754    3.147584    3.908754
 3     0    0.005603   -0.826635    4.125488    4.108379    4.102416    4.108379
 4     0    0.001555   -0.899999    5.087103    5.079159    5.077979    5.079159
 5     0    0.000535   -0.897556    6.067507    6.063251    6.062833    6.063251

 Harris energy:
 sumev=       -9.639146  val*vef=    -181.399258   sumtv=     171.760112
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756520
 rhoeps=    -128.563181     utot=   -6520.515972    ehar=   -3305.562522

 avg es pot at rmt= 0.372049  avg sphere pot= 0.636842  vconst=-0.372049
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           0           5 F

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=F
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===


 ekin=3400.623324  rho*v=-6703.686458 ehf =-3305.562522  ehks =-3303.063134
 mixrho: sum smrho  init = 0.190672D+03-0.545596D-27 0.190672D+03       0
 mixrho: sum smrnew new  = 0.109011D+03 0.364971D-17 0.109011D+03       0
  
 mixrho: dqsum rmsuns= -0.24196D-01  0.24682D-01 -0.20633D-19
 mixrealsmooth= T
 wgtsmooth=  1.72132593164774084E-002
 mixrho:  sought 8 iter from file mixm; read 1.  RMS DQ=1.01e-1  last it=3.44e-2
 charges:       old           new         screened      rms diff       lin mix
 smooth       4.437074      2.536763      2.536763      0.024682      2.536763
 site    1    6.562926      8.463236      8.463236      0.078768      8.463236
 AMIX: nmix=1 mmix=8  nelts=5333  beta=1  tm=5  rmsdel=1.01e-1
   tj: 0.81751
 mixrealsmooth= T
 smrho qcell: add correction to smrho= -1.98727940770027089E-008 -2.53031996933688123E-010
 mixrho: all smrho is positive for isp=           1

 iors  : write restart file (binary, mesh density) 

   it  2  of 20    ehf=   -3305.562522   ehk=   -3303.063134
 From last iter    ehf=   -3304.835818   ehk=   -3304.492675
 diffe(q)= -0.726704 (0.101381)    tol= 0.000010 (0.000010)   more=T
i ehf=-3305.5625216 ehk=-3303.063134

 --- BNDFP:  begin iteration 3 of 20 ---
 ttt nevmx w=           0  5.00000000000000010E-003

 avg es pot at rmt= 0.631894  avg sphere pot= 0.634083  vconst=-0.631894
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           1           5 T

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===
 potential shift to crystal energy zero:    0.000105


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 end of suham2
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.44D-03    3  0.44D-03    4  0.44D-03    5  0.29D-01
    6  0.29D-01    7  0.40D-01    8  0.67D-01    9  0.67D-01   10  0.67D-01
   11  0.31D+00   12  0.31D+00   13  0.31D+00   14  0.36D+00   15  0.46D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7667 -0.5029 -0.5029 -0.5029 -0.4521 -0.4521  1.6438  1.8052  1.8052
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.5792 -0.5061 -0.4873 -0.4843 -0.4551 -0.3795  0.7220  1.1562  1.7106
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.5908 -0.5812 -0.4182 -0.4098 -0.4098  0.0359  0.4030  0.8900  0.8900

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.142459;  11.000000 electrons
         Sum occ. bands:   -5.274909, incl. Bloechl correction: -0.012364

 Saved qp weights ...
 Start second band pass ...
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.44D-03    3  0.44D-03    4  0.44D-03    5  0.29D-01
    6  0.29D-01    7  0.40D-01    8  0.67D-01    9  0.67D-01   10  0.67D-01
   11  0.31D+00   12  0.31D+00   13  0.31D+00   14  0.36D+00   15  0.46D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7667 -0.5029 -0.5029 -0.5029 -0.4521 -0.4521  1.6438  1.8052  1.8052
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.5792 -0.5061 -0.4873 -0.4843 -0.4551 -0.3795  0.7220  1.1562  1.7106
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.5908 -0.5812 -0.4182 -0.4098 -0.4098  0.0359  0.4030  0.8900  0.8900

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.142459;  11.000000 electrons
         Sum occ. bands:   -5.274909, incl. Bloechl correction: -0.012364

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.273018    2.478563    7.794455

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.429288   -0.484088    4.675732    4.658674    4.500000    4.658674
 1     0    0.283944   -0.393195    4.366703    4.366592    4.250000    4.366592
 2     0    9.545093   -0.482273    3.908754    3.892305    3.147584    3.892305
 3     0    0.011079   -0.468597    4.108379    4.115929    4.102416    4.115929
 4     0    0.002780   -0.483678    5.079159    5.082929    5.077979    5.082929
 5     0    0.000835   -0.473242    6.063251    6.065297    6.062833    6.065297

 Harris energy:
 sumev=       -5.274909  val*vef=    -188.047626   sumtv=     182.772718
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756520
 rhoeps=    -129.432876     utot=   -6529.984448    ehar=   -3304.888087

 avg es pot at rmt= 0.477136  avg sphere pot= 0.641278  vconst=-0.477136
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           0           5 F

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=F
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===


 ekin=3376.827204  rho*v=-6681.250065 ehf =-3304.888087  ehks =-3304.422861
 mixrho: sum smrho  init = 0.175769D+03 0.802469D-27 0.175769D+03       0
 mixrho: sum smrnew new  = 0.137750D+03 0.339595D-17 0.137750D+03       0
  
 mixrho: dqsum rmsuns= -0.11265D-01  0.11425D-01 -0.12824D-19
 mixrealsmooth= T
 wgtsmooth=  1.72132593164774084E-002
 mixrho:  sought 8 iter from file mixm; read 2.  RMS DQ=4.17e-2  last it=1.01e-1
 charges:       old           new         screened      rms diff       lin mix
 smooth       4.090277      3.205545      3.205545      0.011425      3.205545
 site    1    6.909723      7.794455      7.794455      0.034207      7.794455
 AMIX: nmix=2 mmix=8  nelts=5333  beta=1  tm=5  rmsdel=4.17e-2
   tj:-0.73043  -0.05116
 mixrealsmooth= T
 smrho qcell: add correction to smrho= -2.40979860954837477E-007 -3.06829604342187205E-009
 mixrho: all smrho is positive for isp=           1

 iors  : write restart file (binary, mesh density) 

   it  3  of 20    ehf=   -3304.888087   ehk=   -3304.422861
 From last iter    ehf=   -3305.562522   ehk=   -3303.063134
 diffe(q)=  0.674435 (0.041697)    tol= 0.000010 (0.000010)   more=T
i ehf=-3304.8880869 ehk=-3304.4228609

 --- BNDFP:  begin iteration 4 of 20 ---
 ttt nevmx w=           0  5.00000000000000010E-003

 avg es pot at rmt= 0.543010  avg sphere pot= 0.644918  vconst=-0.543010
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           1           5 T

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===
 potential shift to crystal energy zero:    0.000100


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 end of suham2
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.56D-03    3  0.56D-03    4  0.56D-03    5  0.39D-01
    6  0.39D-01    7  0.50D-01    8  0.91D-01    9  0.91D-01   10  0.91D-01
   11  0.40D+00   12  0.40D+00   13  0.40D+00   14  0.46D+00   15  0.59D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.6761 -0.0678 -0.0678 -0.0678  0.0065  0.0065  1.6999  1.9075  1.9075
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4300 -0.1174 -0.0677 -0.0436 -0.0000  0.0556  0.8818  1.2855  1.7822
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.2547 -0.1966  0.0552  0.0694  0.0694  0.1313  0.6241  0.9953  0.9953

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.133045;  11.000000 electrons
         Sum occ. bands:   -0.962555, incl. Bloechl correction: -0.006820

 Saved qp weights ...
 Start second band pass ...
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.56D-03    3  0.56D-03    4  0.56D-03    5  0.39D-01
    6  0.39D-01    7  0.50D-01    8  0.91D-01    9  0.91D-01   10  0.91D-01
   11  0.40D+00   12  0.40D+00   13  0.40D+00   14  0.46D+00   15  0.59D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.6761 -0.0678 -0.0678 -0.0678  0.0065  0.0065  1.6999  1.9075  1.9075
 (warning) DOS window (-1,0) reset to (-1.1761,0.6330)
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4300 -0.1174 -0.0677 -0.0436 -0.0000  0.0556  0.8818  1.2855  1.7822
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.2547 -0.1966  0.0552  0.0694  0.0694  0.1313  0.6241  0.9953  0.9953

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.133045;  11.000000 electrons
         Sum occ. bands:   -0.962555, incl. Bloechl correction: -0.006820

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1    9.855952    3.261239    6.594714

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.487308   -0.353357    4.658674    4.656541    4.500000    4.656541
 1     0    0.502065   -0.174962    4.366592    4.396391    4.250000    4.396391
 2     0    8.835915   -0.051917    3.892305    3.868334    3.147584    3.868334
 3     0    0.024398   -0.071741    4.115929    4.125224    4.102416    4.125224
 4     0    0.004985   -0.058075    5.082929    5.086988    5.077979    5.086988
 5     0    0.001282   -0.037146    6.065297    6.067445    6.062833    6.067445

 Harris energy:
 sumev=       -0.962555  val*vef=    -190.404774   sumtv=     189.442219
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756520
 rhoeps=    -130.256522     utot=   -6535.769395    ehar=   -3304.827179

 avg es pot at rmt= 0.683774  avg sphere pot= 0.633966  vconst=-0.683774
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           0           5 F

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=F
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===


 ekin=3345.170049  rho*v=-6649.688679 ehf =-3304.827179  ehks =-3304.518630
 mixrho: sum smrho  init = 0.156035D+03-0.586321D-27 0.156035D+03       0
 mixrho: sum smrnew new  = 0.189306D+03-0.159652D-16 0.189306D+03       0
  
 mixrho: dqsum rmsuns=  0.98581D-02  0.10080D-01 -0.24287D-19
 mixrealsmooth= T
 wgtsmooth=  1.72132593164774084E-002
 mixrho:  sought 8 iter from file mixm; read 3.  RMS DQ=3.25e-2  last it=4.17e-2
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.631043      4.405287      4.405287      0.010080      4.405287
 site    1    7.368957      6.594714      6.594714      0.028507      6.594714
 AMIX: nmix=3 mmix=8  nelts=5333  beta=1  tm=5  rmsdel=3.25e-2
   tj: 0.72976  -0.16331  -0.00000
 mixrealsmooth= T
 smrho qcell: add correction to smrho= -1.27398140925549797E-007 -1.62210738354781012E-009
 mixrho: all smrho is positive for isp=           1

 iors  : write restart file (binary, mesh density) 

   it  4  of 20    ehf=   -3304.827179   ehk=   -3304.518630
 From last iter    ehf=   -3304.888087   ehk=   -3304.422861
 diffe(q)=  0.060908 (0.032470)    tol= 0.000010 (0.000010)   more=T
i ehf=-3304.8271788 ehk=-3304.5186303

 --- BNDFP:  begin iteration 5 of 20 ---
 ttt nevmx w=           0  5.00000000000000010E-003

 avg es pot at rmt= 0.583885  avg sphere pot= 0.638832  vconst=-0.583885
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           1           5 T

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===
 potential shift to crystal energy zero:    0.000102


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 end of suham2
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.78D-01    9  0.78D-01   10  0.78D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7168 -0.2452 -0.2452 -0.2452 -0.1813 -0.1813  1.6704  1.8607  1.8607
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4805 -0.2747 -0.2412 -0.2232 -0.1862 -0.1319  0.8099  1.2287  1.7474
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3845 -0.3509 -0.1392 -0.1278 -0.1278  0.0884  0.5235  0.9479  0.9479

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.009792;  11.000000 electrons
         Sum occ. bands:   -2.696693, incl. Bloechl correction: -0.009350

 Saved qp weights ...
 Start second band pass ...
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.78D-01    9  0.78D-01   10  0.78D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7168 -0.2452 -0.2452 -0.2452 -0.1813 -0.1813  1.6704  1.8607  1.8607
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4805 -0.2747 -0.2412 -0.2232 -0.1862 -0.1319  0.8099  1.2287  1.7474
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3845 -0.3509 -0.1392 -0.1278 -0.1278  0.0884  0.5235  0.9479  0.9479

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.009792;  11.000000 electrons
         Sum occ. bands:   -2.696693, incl. Bloechl correction: -0.009350

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.054917    2.908407    7.146510

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.458594   -0.417172    4.656541    4.654884    4.500000    4.654884
 1     0    0.391469   -0.277780    4.396391    4.380823    4.250000    4.380823
 2     0    9.182226   -0.226726    3.868334    3.878624    3.147584    3.878624
 3     0    0.017608   -0.233836    4.125224    4.121347    4.102416    4.121347
 4     0    0.003938   -0.231284    5.086988    5.085324    5.077979    5.085324
 5     0    0.001082   -0.215596    6.067445    6.066567    6.062833    6.066567

 Harris energy:
 sumev=       -2.696693  val*vef=    -190.146711   sumtv=     187.450018
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756520
 rhoeps=    -129.931782     utot=   -6534.038420    ehar=   -3304.763664

 avg es pot at rmt= 0.587252  avg sphere pot= 0.639004  vconst=-0.587252
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           0           5 F

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=F
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===


 ekin=3358.698119  rho*v=-6663.461505 ehf =-3304.763664  ehks =-3304.763386
 mixrho: sum smrho  init = 0.164795D+03 0.525431D-27 0.164795D+03       0
 mixrho: sum smrnew new  = 0.165594D+03-0.165528D-16 0.165594D+03       0
  
 mixrho: dqsum rmsuns=  0.23659D-03  0.24293D-03 -0.27504D-19
 mixrealsmooth= T
 wgtsmooth=  1.72132593164774084E-002
 mixrho:  sought 8 iter from file mixm; read 4.  RMS DQ=1.00e-3  last it=3.25e-2
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.834908      3.853490      3.853490      0.000243      3.853490
 site    1    7.165092      7.146510      7.146510      0.000851      7.146510
 AMIX: nmix=4 mmix=8  nelts=5333  beta=1  tm=5  rmsdel=1e-3
   tj: 0.39679   0.67043  -0.14970   0.00002
 mixrealsmooth= T
 smrho qcell: add correction to smrho= -1.13463975104366455E-007 -1.44468946286300318E-009
 mixrho: all smrho is positive for isp=           1

 iors  : write restart file (binary, mesh density) 

   it  5  of 20    ehf=   -3304.763664   ehk=   -3304.763386
 From last iter    ehf=   -3304.827179   ehk=   -3304.518630
 diffe(q)=  0.063515 (0.001004)    tol= 0.000010 (0.000010)   more=T
i ehf=-3304.7636643 ehk=-3304.763386

 --- BNDFP:  begin iteration 6 of 20 ---
 ttt nevmx w=           0  5.00000000000000010E-003

 avg es pot at rmt= 0.583943  avg sphere pot= 0.638853  vconst=-0.583943
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           1           5 T

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===
 potential shift to crystal energy zero:    0.000102


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 end of suham2
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7169 -0.2456 -0.2456 -0.2456 -0.1819 -0.1819  1.6709  1.8614  1.8614
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4806 -0.2752 -0.2417 -0.2237 -0.1868 -0.1324  0.8092  1.2279  1.7478
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3849 -0.3513 -0.1398 -0.1283 -0.1283  0.0883  0.5227  0.9481  0.9481

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.010171;  11.000000 electrons
         Sum occ. bands:   -2.701581, incl. Bloechl correction: -0.009352

 Saved qp weights ...
 Start second band pass ...
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7169 -0.2456 -0.2456 -0.2456 -0.1819 -0.1819  1.6709  1.8614  1.8614
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4806 -0.2752 -0.2417 -0.2237 -0.1868 -0.1324  0.8092  1.2279  1.7478
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3849 -0.3513 -0.1398 -0.1283 -0.1283  0.0883  0.5227  0.9481  0.9481

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.010171;  11.000000 electrons
         Sum occ. bands:   -2.701581, incl. Bloechl correction: -0.009352

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.055555    2.907931    7.147624

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.458474   -0.417323    4.654884    4.654880    4.500000    4.654880
 1     0    0.391168   -0.278035    4.380823    4.380782    4.250000    4.380782
 2     0    9.183296   -0.227218    3.878624    3.878650    3.147584    3.878650
 3     0    0.017599   -0.234264    4.121347    4.121336    4.102416    4.121336
 4     0    0.003937   -0.231709    5.085324    5.085320    5.077979    5.085320
 5     0    0.001082   -0.216033    6.066567    6.066565    6.062833    6.066565

 Harris energy:
 sumev=       -2.701581  val*vef=    -190.144398   sumtv=     187.442817
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756520
 rhoeps=    -129.930789     utot=   -6534.032221    ehar=   -3304.763673

 avg es pot at rmt= 0.587023  avg sphere pot= 0.638966  vconst=-0.587023
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           0           5 F

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=F
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===


 ekin=3358.741597  rho*v=-6663.505045 ehf =-3304.763673  ehks =-3304.763449
 mixrho: sum smrho  init = 0.164806D+03-0.121425D-26 0.164806D+03       0
 mixrho: sum smrnew new  = 0.165546D+03-0.170383D-16 0.165546D+03       0
  
 mixrho: dqsum rmsuns=  0.21927D-03  0.22412D-03  0.16343D-19
 mixrealsmooth= T
 wgtsmooth=  1.72132593164774084E-002
 mixrho:  sought 8 iter from file mixm; read 5.  RMS DQ=9.06e-4  last it=1.00e-3
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.835155      3.852376      3.852376      0.000224      3.852376
 site    1    7.164845      7.147624      7.147624      0.000770      7.147624
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=5333  beta=1  tm=5  rmsdel=9.06e-4
   tj:-4.95700  -0.01351
 mixrealsmooth= T
 smrho qcell: add correction to smrho= -2.25884172699863939E-007 -2.87608894212328699E-009
 mixrho: all smrho is positive for isp=           1

 iors  : write restart file (binary, mesh density) 

   it  6  of 20    ehf=   -3304.763673   ehk=   -3304.763449
 From last iter    ehf=   -3304.763664   ehk=   -3304.763386
 diffe(q)= -0.000009 (0.000906)    tol= 0.000010 (0.000010)   more=T
i ehf=-3304.7636733 ehk=-3304.7634487

 --- BNDFP:  begin iteration 7 of 20 ---
 ttt nevmx w=           0  5.00000000000000010E-003

 avg es pot at rmt= 0.584578  avg sphere pot= 0.638845  vconst=-0.584578
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           1           5 T

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===
 potential shift to crystal energy zero:    0.000102


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 end of suham2
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7176 -0.2496 -0.2496 -0.2496 -0.1863 -0.1863  1.6706  1.8606  1.8606
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4816 -0.2787 -0.2457 -0.2278 -0.1910 -0.1367  0.8078  1.2268  1.7472
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3880 -0.3548 -0.1443 -0.1327 -0.1327  0.0876  0.5207  0.9472  0.9472

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.012904;  11.000000 electrons
         Sum occ. bands:   -2.741362, incl. Bloechl correction: -0.009395

 Saved qp weights ...
 Start second band pass ...
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7176 -0.2496 -0.2496 -0.2496 -0.1863 -0.1863  1.6706  1.8606  1.8606
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4816 -0.2787 -0.2457 -0.2278 -0.1910 -0.1367  0.8078  1.2268  1.7472
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3880 -0.3548 -0.1443 -0.1327 -0.1327  0.0876  0.5207  0.9472  0.9472

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.012904;  11.000000 electrons
         Sum occ. bands:   -2.741362, incl. Bloechl correction: -0.009395

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.059917    2.899949    7.159968

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.457782   -0.418517    4.654880    4.654867    4.500000    4.654867
 1     0    0.388860   -0.280075    4.380782    4.380472    4.250000    4.380472
 2     0    9.190824   -0.231214    3.878650    3.878892    3.147584    3.878892
 3     0    0.017461   -0.237952    4.121336    4.121245    4.102416    4.121245
 4     0    0.003914   -0.235658    5.085320    5.085281    5.077979    5.085281
 5     0    0.001077   -0.220088    6.066565    6.066544    6.062833    6.066544

 Harris energy:
 sumev=       -2.741362  val*vef=    -190.126130   sumtv=     187.384768
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756520
 rhoeps=    -129.922720     utot=   -6533.982258    ehar=   -3304.763692

 avg es pot at rmt= 0.584880  avg sphere pot= 0.639044  vconst=-0.584880
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           0           5 F

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=F
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===


 ekin=3359.063089  rho*v=-6663.826775 ehf =-3304.763692  ehks =-3304.763686
 mixrho: sum smrho  init = 0.164988D+03-0.918037D-28 0.164988D+03       0
 mixrho: sum smrnew new  = 0.165016D+03 0.140913D-16 0.165016D+03       0
  
 mixrho: dqsum rmsuns=  0.82062D-05  0.48422D-04 -0.51378D-19
 mixrealsmooth= T
 wgtsmooth=  1.72132593164774084E-002
 mixrho:  sought 8 iter from file mixm; read 6.  RMS DQ=1.47e-4  last it=9.06e-4
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.839387      3.840032      3.840032      0.000048      3.840032
 site    1    7.160613      7.159968      7.159968      0.000126      7.159968
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=5333  beta=1  tm=5  rmsdel=1.47e-4
   tj:-0.17342
 mixrealsmooth= T
 smrho qcell: add correction to smrho= -3.15239923054377869E-009 -4.01381843612913807E-011
 mixrho: all smrho is positive for isp=           1

 iors  : write restart file (binary, mesh density) 

   it  7  of 20    ehf=   -3304.763692   ehk=   -3304.763686
 From last iter    ehf=   -3304.763673   ehk=   -3304.763449
 diffe(q)= -0.000018 (0.000147)    tol= 0.000010 (0.000010)   more=T
i ehf=-3304.7636917 ehk=-3304.7636859

 --- BNDFP:  begin iteration 8 of 20 ---
 ttt nevmx w=           0  5.00000000000000010E-003

 avg es pot at rmt= 0.584508  avg sphere pot= 0.639058  vconst=-0.584508
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           1           5 T

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===
 potential shift to crystal energy zero:    0.000102


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 end of suham2
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7178 -0.2506 -0.2506 -0.2506 -0.1874 -0.1874  1.6705  1.8604  1.8604
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4819 -0.2796 -0.2467 -0.2289 -0.1921 -0.1377  0.8074  1.2266  1.7471
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3887 -0.3557 -0.1454 -0.1339 -0.1339  0.0874  0.5201  0.9470  0.9470

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013582;  11.000000 electrons
         Sum occ. bands:   -2.751237, incl. Bloechl correction: -0.009405

 Saved qp weights ...
 Start second band pass ...
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7178 -0.2506 -0.2506 -0.2506 -0.1874 -0.1874  1.6705  1.8604  1.8604
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4819 -0.2796 -0.2467 -0.2289 -0.1921 -0.1377  0.8074  1.2266  1.7471
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3887 -0.3557 -0.1454 -0.1339 -0.1339  0.0874  0.5201  0.9470  0.9470

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013582;  11.000000 electrons
         Sum occ. bands:   -2.751237, incl. Bloechl correction: -0.009405

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.060997    2.897854    7.163142

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.457600   -0.418812    4.654867    4.654848    4.500000    4.654848
 1     0    0.388282   -0.280583    4.380472    4.380380    4.250000    4.380380
 2     0    9.192703   -0.232205    3.878892    3.878950    3.147584    3.878950
 3     0    0.017427   -0.238868    4.121245    4.121220    4.102416    4.121220
 4     0    0.003908   -0.236641    5.085281    5.085270    5.077979    5.085270
 5     0    0.001076   -0.221096    6.066544    6.066538    6.062833    6.066538

 Harris energy:
 sumev=       -2.751237  val*vef=    -190.113560   sumtv=     187.362323
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756520
 rhoeps=    -129.920581     utot=   -6533.961971    ehar=   -3304.763709

 avg es pot at rmt= 0.584338  avg sphere pot= 0.639065  vconst=-0.584338
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           0           5 F

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=F
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===


 ekin=3359.146492  rho*v=-6663.910200 ehf =-3304.763709  ehks =-3304.763708
 mixrho: sum smrho  init = 0.164924D+03 0.170591D-27 0.164924D+03       0
 mixrho: sum smrnew new  = 0.164879D+03 0.756802D-17 0.164879D+03       0
  
 mixrho: dqsum rmsuns= -0.13157D-04  0.13410D-04  0.10913D-19
 mixrealsmooth= T
 wgtsmooth=  1.72132593164774084E-002
 mixrho:  sought 8 iter from file mixm; read 7.  RMS DQ=5.08e-5  last it=1.47e-4
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.837891      3.836858      3.836858      0.000013      3.836858
 site    1    7.162109      7.163142      7.163142      0.000040      7.163142
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=5333  beta=1  tm=5  rmsdel=5.08e-5
   tj: 0.03869   0.04471
 mixrealsmooth= T
 smrho qcell: add correction to smrho= -4.10325728950056146E-009 -5.22450633701545939E-011
 mixrho: all smrho is positive for isp=           1

 iors  : write restart file (binary, mesh density) 

   it  8  of 20    ehf=   -3304.763709   ehk=   -3304.763708
 From last iter    ehf=   -3304.763692   ehk=   -3304.763686
 diffe(q)= -0.000017 (0.000051)    tol= 0.000010 (0.000010)   more=T
i ehf=-3304.7637088 ehk=-3304.7637081

 --- BNDFP:  begin iteration 9 of 20 ---
 ttt nevmx w=           0  5.00000000000000010E-003

 avg es pot at rmt= 0.584479  avg sphere pot= 0.639060  vconst=-0.584479
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           1           5 T

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===
 potential shift to crystal energy zero:    0.000102


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 end of suham2
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7177 -0.2504 -0.2504 -0.2504 -0.1871 -0.1871  1.6705  1.8605  1.8605
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4818 -0.2794 -0.2465 -0.2287 -0.1919 -0.1375  0.8075  1.2266  1.7471
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3886 -0.3555 -0.1452 -0.1336 -0.1336  0.0874  0.5202  0.9471  0.9471

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013446;  11.000000 electrons
         Sum occ. bands:   -2.749249, incl. Bloechl correction: -0.009403

 Saved qp weights ...
 Start second band pass ...
 -------- qplist --------
    1   0.000   0.000   0.000
    2  -0.125   0.125   0.125
    3  -0.250   0.250   0.250
    4  -0.375   0.375   0.375
    5  -0.500   0.500   0.500
    6   0.000   0.000   0.250
    7  -0.125   0.125   0.375
    8  -0.250   0.250   0.500
    9  -0.375   0.375   0.625
   10  -0.500   0.500   0.750
   11  -0.625   0.625   0.875
   12  -0.750   0.750   1.000
   13   0.000   0.000   0.500
   14  -0.125   0.125   0.625
   15  -0.250   0.250   0.750
   16  -0.375   0.375   0.875
   17  -0.500   0.500   1.000
   18   0.000   0.000   0.750
   19  -0.125   0.125   0.875
   20  -0.250   0.250   1.000
   21   0.000   0.000   1.000
   22   0.000   0.250   0.500
   23  -0.125   0.375   0.625
   24  -0.250   0.500   0.750
   25   0.000   0.250   0.750
   26  -0.125   0.375   0.875
   27  -0.250   0.500   1.000
   28   0.000   0.250   1.000
   29   0.000   0.500   1.000
 sigmamode= F
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   ... skip larger eigenvalues ...
 eigenvalue=
 -0.7177 -0.2504 -0.2504 -0.2504 -0.1871 -0.1871  1.6705  1.8605  1.8605
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4818 -0.2794 -0.2465 -0.2287 -0.1919 -0.1375  0.8075  1.2266  1.7471
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3886 -0.3555 -0.1452 -0.1336 -0.1336  0.0874  0.5202  0.9471  0.9471

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013446;  11.000000 electrons
         Sum occ. bands:   -2.749249, incl. Bloechl correction: -0.009403

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.060772    2.898298    7.162474

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.457637   -0.418756    4.654848    4.654851    4.500000    4.654851
 1     0    0.388401   -0.280483    4.380380    4.380397    4.250000    4.380397
 2     0    9.192313   -0.232005    3.878950    3.878938    3.147584    3.878938
 3     0    0.017434   -0.238683    4.121220    4.121225    4.102416    4.121225
 4     0    0.003910   -0.236443    5.085270    5.085272    5.077979    5.085272
 5     0    0.001076   -0.220892    6.066538    6.066539    6.062833    6.066539

 Harris energy:
 sumev=       -2.749249  val*vef=    -190.117889   sumtv=     187.368641
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756520
 rhoeps=    -129.921067     utot=   -6533.967798    ehar=   -3304.763705

 avg es pot at rmt= 0.584452  avg sphere pot= 0.639062  vconst=-0.584452
 smvxcm: all smrho_w is positive
  i job kmax lfltwf(FRZWF see locpot.F)=           0           0           5 F

 site  1  z= 29.0  rmt= 2.28000  nr=655   a=0.015  nlml=36  rg=0.570  Vfloat=F
 sm core charge = 0.295521 (sphere) + 0.00535 (spillout) = 0.30087
 === rho1 valence true density ===
 === rho2 valence counter density ===
 === rhol1 valence+core density ===
 === rho2 ->valence+smooth core density ===


 ekin=3359.128252  rho*v=-6663.891956 ehf =-3304.763705  ehks =-3304.763705
 mixrho: sum smrho  init = 0.164914D+03-0.199779D-27 0.164914D+03       0
 mixrho: sum smrnew new  = 0.164908D+03 0.181775D-17 0.164908D+03       0
  
 mixrho: dqsum rmsuns= -0.18874D-05  0.19484D-05  0.41811D-19
 mixrealsmooth= T
 wgtsmooth=  1.72132593164774084E-002
 mixrho:  sought 8 iter from file mixm; read 8.  RMS DQ=6.36e-6  last it=5.08e-5
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.837674      3.837526      3.837526      0.000002      3.837526
 site    1    7.162326      7.162474      7.162474      0.000006      7.162474
 AMIX: condition of normal eqns >100000. Reducing nmix to 7
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=5333  beta=1  tm=5  rmsdel=6.36e-6
   tj:-0.13742
 mixrealsmooth= T
 smrho qcell: add correction to smrho= -4.48256542995295604E-009 -5.70746356919923363E-011
 mixrho: all smrho is positive for isp=           1

 iors  : write restart file (binary, mesh density) 

   it  9  of 20    ehf=   -3304.763705   ehk=   -3304.763705
 From last iter    ehf=   -3304.763709   ehk=   -3304.763708
 diffe(q)=  0.000004 (0.000006)    tol= 0.000010 (0.000010)   more=F
c ehf=-3304.7637048 ehk=-3304.7637047
 Exit 0 LMF 
