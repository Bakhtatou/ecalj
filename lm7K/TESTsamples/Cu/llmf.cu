#include "hip/hip_runtime.h"
 -----------------------  START LMF (80000K)  -----------------------
 HEADER Cu

 rdctrl: reset global max nl from 5 to 6

 LMF:      alat = 6.798  nbas = 1  nspec = 1  vn 7.00(LMF 7.0)  verb 31,20
 special:  APW basis
 pot:      XC:BH
 bz:       metal(3), tetra, invit 

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-08   alat= 6.79800   awald= 0.467
         r1=  1.959   nkd= 135      q1=  5.910   nkg= 181

 SGROUP: 1 symmetry operations from 0 generators
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 GROUPG: the following are sufficient to generate the space group:
         i*r3(1,1,-1) r4x
         i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 
 BZMESH:  29 irreducible QP from 512 ( 8 8 8 )  shift= F F F
 TETIRR: sorting 3072 tetrahedra ...
 76 inequivalent ones found

 species data:  augmentation                           density
 spec       rmt   rsma lmxa kmxa      lmxl     rg   rsmv  kmxv foca   rfoca
 Cu       2.280  0.912    5    5         5  0.570  1.140    15    1   0.912

 gvlist: cutoff radius   9.803 gives   1243   recips of max   3375
 SGVSYM: 53 symmetry stars found for 1243 reciprocal lattice vectors
 

 Makidx:  hamiltonian dimensions Low, Int, High, Negl: 9 0 27 0
 suham :  36 augmentation channels, 36 local potential channels  Maximum lmxa=5

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  Cu       0    1.50  -0.28   4.956    2.32E-06     169 
  Cu       1    1.50  -0.10   5.245    1.10E-06     181 
  Cu       2    0.95  -0.11   8.973    1.81E-06     941 

 suham:  q-dependent PW basis with  Emin = 0 < E < 5.
         Est. min,max PW dimension = 12,18.  Use npwpad = 3 => ndham = 30

 iors  : read restart file (binary, mesh density) 
 iors  : empty file ... nothing read

 rdovfa: read and overlap free-atom densities (mesh density) ...
 rdovfa: expected Cu,      read Cu       with rmt=  2.2800  mesh   393  0.025

 ovlpfa: overlap smooth part of FA densities
 site   1  spec  1  pos  0.0000  0.0000  0.0000  Qsmooth 4.729619
 total smooth Q = 4.729619

 Free atom and overlapped crystal site charges:
   ib    true(FA)    smooth(FA)  true(OV)    smooth(OV)    local
    1    9.758197    3.487816   10.209423    3.939042    6.270381

 Smooth charge on mesh:            4.729619
 Sum of local charges:             6.270381
 Total valence charge:            11.000000
 Sum of core charges:             18.000000
 Sum of nuclear charges:         -29.000000
 Homogeneous background:           0.000000
 Deviation from neutrality:        0.000000

 --- BNDFP:  begin iteration 1 of 20 ---

 avg es pot at rmt= 0.554330  avg sphere pot= 0.625056  vconst=-0.554330

 site  1  z= 29.0  rmt= 2.28000  nr=393   a=0.025  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295519 (sphere) + 0.00535 (spillout) = 0.300868
 potential shift to crystal energy zero:    0.000086


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.95D-06    2  0.57D-03    3  0.57D-03    4  0.57D-03    5  0.40D-01
    6  0.40D-01    7  0.51D-01    8  0.90D-01    9  0.90D-01   10  0.90D-01
   11  0.40D+00   12  0.40D+00   13  0.40D+00   14  0.47D+00   15  0.59D+00
   16  0.10D+05   17  0.10D+05   18  0.10D+05   19  0.10D+05   20  0.10D+05
   21  0.10D+05   22  0.10D+05   23  0.10D+05   24  0.10D+05
 eigenvalue=
 -0.6650 -0.0472 -0.0472 -0.0472  0.0270  0.0270  1.7100  1.9162  1.9162
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4185 -0.0983 -0.0478 -0.0233  0.0207  0.0764  0.8953  1.2978  1.7913
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.2385 -0.1775  0.0760  0.0911  0.0911  0.1426  0.6393  1.0060  1.0060

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.151736;  11.000000 electrons
         Sum occ. bands:   -0.756621, incl. Bloechl correction: -0.006665

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.95D-06    2  0.57D-03    3  0.57D-03    4  0.57D-03    5  0.40D-01
    6  0.40D-01    7  0.51D-01    8  0.90D-01    9  0.90D-01   10  0.90D-01
   11  0.40D+00   12  0.40D+00   13  0.40D+00   14  0.47D+00   15  0.59D+00
   16  0.10D+05   17  0.10D+05   18  0.10D+05   19  0.10D+05   20  0.10D+05
   21  0.10D+05   22  0.10D+05   23  0.10D+05   24  0.10D+05
 eigenvalue=
 -0.6650 -0.0472 -0.0472 -0.0472  0.0270  0.0270  1.7100  1.9162  1.9162
 (warning) DOS window (-1,0) reset to (-1.1650,0.6517)
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4185 -0.0983 -0.0478 -0.0233  0.0207  0.0764  0.8953  1.2978  1.7913
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.2385 -0.1775  0.0760  0.0911  0.0911  0.1426  0.6393  1.0060  1.0060

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.151736;  11.000000 electrons
         Sum occ. bands:   -0.756621, incl. Bloechl correction: -0.006665

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1    9.841193    3.278284    6.562910

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.488976   -0.340549    4.690000    4.656558    4.500000    4.656558
 1     0    0.510458   -0.159503    4.420000    4.397640    4.250000    4.397640
 2     0    8.810553   -0.031904    3.880000    3.867564    3.147584    3.867564
 3     0    0.024903   -0.052671    4.120000    4.125488    4.102416    4.125488
 4     0    0.005037   -0.038215    5.100000    5.087104    5.077979    5.087104
 5     0    0.001265   -0.016835    6.100000    6.067507    6.062833    6.067507

 Harris energy:
 sumev=       -0.756621  val*vef=    -189.393464   sumtv=     188.636843
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -130.258100     utot=   -6534.971246    ehar=   -3304.835864

 ekin=3344.586423  rho*v=-6649.079139  ehf=-3304.835864  ehks=-3304.492716
  
 mixrho:  sought 8 iter from file mixm; read 8.  RMS DQ=3.77e-2
 charges:       old           new         screened      rms diff       lin mix
 smooth       4.729619      4.437091      4.437091      0.045494      4.437091
 site    1    6.270381      6.562910      6.562910      0.014896      6.562910
 AMIX: condition of normal eqns >100000. Reducing nmix to 7
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=2678  beta=1  tm=5  rmsdel=3.77e-2
   tj: 1.00000
 unscreened rms difference:  smooth  0.045494   local  0.014896

 iors  : write restart file (binary, mesh density) 

   it  1  of 20    ehf=   -3304.835864   ehk=   -3304.492716
h ehf=-3304.8358636 ehk=-3304.4927165

 --- BNDFP:  begin iteration 2 of 20 ---

 avg es pot at rmt= 0.584471  avg sphere pot= 0.639061  vconst=-0.584471

 site  1  z= 29.0  rmt= 2.28000  nr=393   a=0.025  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295519 (sphere) + 0.00535 (spillout) = 0.300868
 potential shift to crystal energy zero:    0.000102


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.34D-01
    6  0.34D-01    7  0.45D-01    8  0.78D-01    9  0.78D-01   10  0.78D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   16  0.10D+05   17  0.10D+05   18  0.10D+05   19  0.10D+05   20  0.10D+05
   21  0.10D+05   22  0.10D+05   23  0.10D+05   24  0.10D+05
 eigenvalue=
 -0.7177 -0.2504 -0.2504 -0.2504 -0.1871 -0.1871  1.6698  1.8595  1.8595
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4818 -0.2794 -0.2464 -0.2286 -0.1918 -0.1374  0.8081  1.2272  1.7467
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3885 -0.3555 -0.1451 -0.1336 -0.1336  0.0874  0.5208  0.9468  0.9468

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013372;  11.000000 electrons
         Sum occ. bands:   -2.748811, incl. Bloechl correction: -0.009406

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.34D-01
    6  0.34D-01    7  0.45D-01    8  0.78D-01    9  0.78D-01   10  0.78D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   16  0.10D+05   17  0.10D+05   18  0.10D+05   19  0.10D+05   20  0.10D+05
   21  0.10D+05   22  0.10D+05   23  0.10D+05   24  0.10D+05
 eigenvalue=
 -0.7177 -0.2504 -0.2504 -0.2504 -0.1871 -0.1871  1.6698  1.8595  1.8595
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4818 -0.2794 -0.2464 -0.2286 -0.1918 -0.1374  0.8081  1.2272  1.7467
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3885 -0.3555 -0.1451 -0.1336 -0.1336  0.0874  0.5208  0.9468  0.9468

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013372;  11.000000 electrons
         Sum occ. bands:   -2.748811, incl. Bloechl correction: -0.009406

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.060610    2.897891    7.162718

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.457684   -0.418743    4.656558    4.654853    4.500000    4.654853
 1     0    0.388448   -0.280459    4.397640    4.380403    4.250000    4.380403
 2     0    9.192066   -0.231959    3.867564    3.878939    3.147584    3.878939
 3     0    0.017427   -0.238670    4.125488    4.121226    4.102416    4.121226
 4     0    0.003909   -0.236466    5.087104    5.085272    5.077979    5.085272
 5     0    0.001076   -0.220914    6.067507    6.066539    6.062833    6.066539

 Harris energy:
 sumev=       -2.748811  val*vef=    -190.117846   sumtv=     187.369035
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.921166     utot=   -6533.968244    ehar=   -3304.763735

 ekin=3359.119828  rho*v=-6663.883561  ehf=-3304.763735  ehks=-3304.763733
  
 mixrho:  sought 8 iter from file mixm; read 8.  RMS DQ=1.73e-5  last it=3.77e-2
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.837641      3.837281      3.837281      0.000011      3.837281
 site    1    7.162359      7.162718      7.162718      0.000020      7.162718
 AMIX: condition of normal eqns >100000. Reducing nmix to 7
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=2678  beta=1  tm=5  rmsdel=1.73e-5
   tj:-0.00022
 unscreened rms difference:  smooth  0.000011   local  0.000020

 iors  : write restart file (binary, mesh density) 

   it  2  of 20    ehf=   -3304.763735   ehk=   -3304.763733
 From last iter    ehf=   -3304.835864   ehk=   -3304.492716
 diffe(q)=  0.072128 (0.000017)    tol= 0.000010 (0.000010)   more=T
i ehf=-3304.7637353 ehk=-3304.7637328

 --- BNDFP:  begin iteration 3 of 20 ---

 avg es pot at rmt= 0.584421  avg sphere pot= 0.639112  vconst=-0.584421

 site  1  z= 29.0  rmt= 2.28000  nr=393   a=0.025  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295519 (sphere) + 0.00535 (spillout) = 0.300868
 potential shift to crystal energy zero:    0.000102


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   16  0.10D+05   17  0.10D+05   18  0.10D+05   19  0.10D+05   20  0.10D+05
   21  0.10D+05   22  0.10D+05   23  0.10D+05   24  0.10D+05
 eigenvalue=
 -0.7177 -0.2504 -0.2504 -0.2504 -0.1872 -0.1872  1.6705  1.8605  1.8605
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4818 -0.2794 -0.2465 -0.2287 -0.1919 -0.1375  0.8075  1.2266  1.7471
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3886 -0.3555 -0.1452 -0.1336 -0.1336  0.0874  0.5202  0.9471  0.9471

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013455;  11.000000 electrons
         Sum occ. bands:   -2.749399, incl. Bloechl correction: -0.009403

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   16  0.10D+05   17  0.10D+05   18  0.10D+05   19  0.10D+05   20  0.10D+05
   21  0.10D+05   22  0.10D+05   23  0.10D+05   24  0.10D+05
 eigenvalue=
 -0.7177 -0.2504 -0.2504 -0.2504 -0.1872 -0.1872  1.6705  1.8605  1.8605
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4818 -0.2794 -0.2465 -0.2287 -0.1919 -0.1375  0.8075  1.2266  1.7471
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3886 -0.3555 -0.1452 -0.1336 -0.1336  0.0874  0.5202  0.9471  0.9471

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013455;  11.000000 electrons
         Sum occ. bands:   -2.749399, incl. Bloechl correction: -0.009403

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.060799    2.898227    7.162572

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.457632   -0.418755    4.654853    4.654848    4.500000    4.654848
 1     0    0.388387   -0.280487    4.380403    4.380393    4.250000    4.380393
 2     0    9.192361   -0.232021    3.878939    3.878939    3.147584    3.878939
 3     0    0.017434   -0.238697    4.121226    4.121224    4.102416    4.121224
 4     0    0.003909   -0.236459    5.085272    5.085272    5.077979    5.085272
 5     0    0.001076   -0.220909    6.066539    6.066539    6.062833    6.066539

 Harris energy:
 sumev=       -2.749399  val*vef=    -190.115850   sumtv=     187.366451
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.920980     utot=   -6533.965861    ehar=   -3304.763750

 ekin=3359.131366  rho*v=-6663.895117  ehf=-3304.763750  ehks=-3304.763750
  
 mixrho:  sought 8 iter from file mixm; read 8.  RMS DQ=2.03e-5  last it=1.73e-5
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.837147      3.837428      3.837428      0.000010      3.837428
 site    1    7.162853      7.162572      7.162572      0.000021      7.162572
 AMIX: condition of normal eqns >100000. Reducing nmix to 7
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=2678  beta=1  tm=5  rmsdel=2.03e-5
   tj: 0.54081
 unscreened rms difference:  smooth  0.000010   local  0.000021

 iors  : write restart file (binary, mesh density) 

   it  3  of 20    ehf=   -3304.763750   ehk=   -3304.763750
 From last iter    ehf=   -3304.763735   ehk=   -3304.763733
 diffe(q)= -0.000015 (0.000020)    tol= 0.000010 (0.000010)   more=T
i ehf=-3304.7637504 ehk=-3304.7637503

 --- BNDFP:  begin iteration 4 of 20 ---

 avg es pot at rmt= 0.584441  avg sphere pot= 0.639088  vconst=-0.584441

 site  1  z= 29.0  rmt= 2.28000  nr=393   a=0.025  nlml=36  rg=0.570  Vfloat=T
 sm core charge = 0.295519 (sphere) + 0.00535 (spillout) = 0.300868
 potential shift to crystal energy zero:    0.000102


 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   16  0.10D+05   17  0.10D+05   18  0.10D+05   19  0.10D+05   20  0.10D+05
   21  0.10D+05   22  0.10D+05   23  0.10D+05   24  0.10D+05
 eigenvalue=
 -0.7177 -0.2504 -0.2504 -0.2504 -0.1871 -0.1871  1.6705  1.8605  1.8605
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4818 -0.2794 -0.2464 -0.2286 -0.1918 -0.1375  0.8075  1.2266  1.7471
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3885 -0.3554 -0.1452 -0.1336 -0.1336  0.0874  0.5203  0.9471  0.9471

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013421;  11.000000 electrons
         Sum occ. bands:   -2.748892, incl. Bloechl correction: -0.009402

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 29, k=  0.00000  0.00000  0.00000   ndimh = 24
 zhev_tk: ovlmat=
    1  0.94D-06    2  0.51D-03    3  0.51D-03    4  0.51D-03    5  0.35D-01
    6  0.35D-01    7  0.45D-01    8  0.79D-01    9  0.79D-01   10  0.79D-01
   11  0.36D+00   12  0.36D+00   13  0.36D+00   14  0.42D+00   15  0.53D+00
   16  0.10D+05   17  0.10D+05   18  0.10D+05   19  0.10D+05   20  0.10D+05
   21  0.10D+05   22  0.10D+05   23  0.10D+05   24  0.10D+05
 eigenvalue=
 -0.7177 -0.2504 -0.2504 -0.2504 -0.1871 -0.1871  1.6705  1.8605  1.8605
 bndfp:  kpt 11 of 29, k=  0.37500  -0.37500  -0.12500   ndimh = 25
 -0.4818 -0.2794 -0.2464 -0.2286 -0.1918 -0.1375  0.8075  1.2266  1.7471
 bndfp:  kpt 21 of 29, k=  0.00000  0.00000  -1.00000   ndimh = 23
 -0.3885 -0.3554 -0.1452 -0.1336 -0.1336  0.0874  0.5203  0.9471  0.9471

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.013421;  11.000000 electrons
         Sum occ. bands:   -2.748892, incl. Bloechl correction: -0.009402

 Saved qp weights ...

 mkrout:  Qtrue      sm,loc       local
   1   10.060738    2.898358    7.162380

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:Cu      
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.457643   -0.418743    4.654848    4.654850    4.500000    4.654850
 1     0    0.388420   -0.280463    4.380393    4.380399    4.250000    4.380399
 2     0    9.192253   -0.231969    3.878939    3.878936    3.147584    3.878936
 3     0    0.017436   -0.238650    4.121224    4.121226    4.102416    4.121226
 4     0    0.003910   -0.236408    5.085272    5.085272    5.077979    5.085272
 5     0    0.001076   -0.220857    6.066539    6.066540    6.062833    6.066540

 Harris energy:
 sumev=       -2.748892  val*vef=    -190.117371   sumtv=     187.368479
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.921120     utot=   -6533.967748    ehar=   -3304.763749

 ekin=3359.126145  rho*v=-6663.889894  ehf=-3304.763749  ehks=-3304.763749
  
 mixrho:  sought 8 iter from file mixm; read 8.  RMS DQ=7.93e-6  last it=2.03e-5
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.837349      3.837620      3.837620      0.000007      3.837620
 site    1    7.162651      7.162380      7.162380      0.000011      7.162380
 AMIX: condition of normal eqns >100000. Reducing nmix to 7
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=2678  beta=1  tm=5  rmsdel=7.93e-6
   tj:-0.28582
 unscreened rms difference:  smooth  0.000007   local  0.000011

 iors  : write restart file (binary, mesh density) 

   it  4  of 20    ehf=   -3304.763749   ehk=   -3304.763749
 From last iter    ehf=   -3304.763750   ehk=   -3304.763750
 diffe(q)=  0.000001 (0.000008)    tol= 0.000010 (0.000010)   more=F
c ehf=-3304.7637492 ehk=-3304.7637492
 Exit 0 LMF 
 wkinfo:  used  1003 K  workspace of 80000 K   in  72 K calls
