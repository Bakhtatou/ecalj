#include "hip/hip_runtime.h"
HOST_INFORMATION platform: gfortran
HOST_INFORMATION compiler version: gcc バージョン 4.6.1 (Ubuntu/Linaro 4.6.1-9ubuntu3) 
HOST_INFORMATION FFLAGS (<=120): -O2 -fomit-frame-pointer -funroll-loops -ffast-math -ffixed-line-length-132 -DHASIARGC -DHASGETARG -DFDATE -DHASCPUTIME 
HOST_INFORMATION LIBLOC (<=120): /usr/lib/libfftw3.so.3 /usr/lib/liblapack.so.3gf /usr/lib/libblas.so.3gf
HOST_INFORMATION uname -a (<=120): Linux TT4 3.0.0-20-generic #34-Ubuntu SMP Tue May 1 17:24:39 UTC 2012 x86_64 x86_64 x86_64 GNU/Linux
HOST_INFORMATION /etc/issue: Ubuntu 11.10 \n \l
HOST_INFORMATION git branch: refs/heads/newaniso
HOST_INFORMATION git commit: ff4f0b57505a84e0eaabe44d640190810c88782c
HOST_INFORMATION linked at: Tue Jun 19 14:13:38 JST 2012
 -----------------------  START LMFA     -----------------------
 ptenv() is called with EXT=cu
 ptenv() not supported, but continue.
 HEADER Cu
 Cu       xxx            1           1

 rdctrl: reset global max nl from 5 to 6
  mxcst switch =           1           0 F F F
  LMFA  vn 7.00(LMFA 7.0)  verb 31,20
 end of rdctrl2 in imfav7
 lattic:

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-08   alat= 6.79800   awald= 0.467
         r1=  1.959   nkd= 135      q1=  5.910   nkg= 181
 goto mksym

 SGROUP: 1 symmetry operations from 0 generators
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 GROUPG: the following are sufficient to generate the space group:
         i*r3(1,1,-1) r4x
         i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 zzz nclass=           1
 end of mksym x
 goto defspc
 end of defspc
 goto freeat

ttt: pnu qat=  1  0     4.690     1.000
ttt: pnu qat=  1  1     4.420     0.000
ttt: pnu qat=  1  2     3.880    10.000
ttt: pnu qat=  1  3     4.120     0.000
ttt: pnu qat=  1  4     5.100     0.000
ttt: pnu qat=  1  5     6.100     0.000
 NOTE: when we have two valence: P and PZ, We assume eigen(PZ) is deeper than eigen(P).
=== Charge for l     : Qtot=Qv=     0   1.000   1.000
=== Charge for l     : Qtot=Qv=     1   0.000   0.000
=== Charge for l     : Qtot=Qv=     2  10.000  10.000
=== Charge for l     : Qtot=Qv=     3   0.000   0.000
=== Charge for l     : Qtot=Qv=     4   0.000   0.000
=== Charge for l     : Qtot=Qv=     5   0.000   0.000

conf:SPEC_ATOM= Cu : --- Table for atomic configuration ---
conf When int(P)z .ne. int(P), Qval: Q for MTOcore(PZ)+MTO(P)
conf:  isp  l  int(P) int(P)z    Qval     Qcore   CoreConf
conf:    1  0       4  4         1.000    6.000 => 1,2,3,
conf:    1  1       4  4         0.000   12.000 => 2,3,
conf:    1  2       3  3        10.000    0.000 => 
conf:    1  3       4  4         0.000    0.000 => 
conf:    1  4       5  5         0.000    0.000 => 
conf:    1  5       6  6         0.000    0.000 => 
conf:-----------------------------------------------------

 Species Cu:  Z=29  Qc=18  R=2.280000  Q=0
 mesh:   rmt=2.280000  rmax=48.629375  a=0.015  nr=655  nr(rmax)=859

  iter     qint         drho          vh0          rho0          vsum     beta
 NOTE: rhocor: core density is spin-independent now for any MMOM, june2012.
       We use spin-avaraged potential to calculate core density rhoc.
       Thus diff of core eigen is pot. due to valence electron. I/O:iofa.F
    1   29.000000   7.875E+03      145.0000    0.1442E+03      -58.3040   0.30
   52   29.000000   4.902E-05      274.8263    0.2697E+05     -130.8188   0.30


 sumev=-4.333259  etot=-3304.416215  eref=0.000000

 Free-atom wavefunctions:
 valence:      eval       node at      max at       c.t.p.   rho(r>rmt)
   4s      -0.36411         0.890       2.256       3.581     0.655341
   4p      -0.06295         0.975       3.484       7.412     0.906557
   3d      -0.39692         0.000       0.600       3.428     0.058644
   4f       0.01964         0.000      35.282      48.629*    1.000000
   5g       0.02702         0.000      36.739      48.629*    1.000000
   65       0.03539         0.000      37.851      48.629*    1.000000

 core:        ecore       node at      max at       c.t.p.   rho(r>rmt)
   1s    -649.07635         0.000       0.034       0.069     0.000000
   2s     -77.91381         0.070       0.197       0.308     0.000000
   2p     -67.32533         0.000       0.158       0.335     0.000000
   3s      -8.39248         0.288       0.614       0.895     0.000167
   3p      -5.29682         0.260       0.619       1.078     0.000836

 Optimise free-atom basis for species Cu, rmt=2.28
 l  it    Rsm      Eh     stiffR   stiffE      Eval      Exact     Pnu    Ql
 0  10   1.140  -0.104       0.0   6532.8   -0.34064  -0.36411    4.75   1.00
 1  11   1.140  -0.100       0.0  -3648.1    0.18334  -0.06295    4.55   0.00
 2   7   1.140  -0.497       0.0     21.3   -0.39473  -0.39692    3.89  10.00
 l=4  increase Pnu=   5.091  to    5.100
 l=5  increase Pnu=   6.069  to    6.100
 eigenvalue sum:  exact  -4.33326    opt basis  -4.28791    error 0.04535
 tailsm: init

 tailsm: fit tails to 6 smoothed hankels, rmt= 2.28000, rsm= 1.14000
  ---E:energies of smHankels. C:fitting coeeficient for core tail. ---
 E:    -1.00000    -2.00000    -4.00000    -6.00000    -9.00000    -15.0000
 C:    -0.07224    10.68573    -181.432    1154.646    -4317.83    18314.70
        r          rho         fit         diff
    2.280000    0.018887    0.018853    0.000034
    2.649002    0.009672    0.009671    0.000002
    3.077722    0.004737    0.004737    0.000000
    3.575823    0.002169    0.002173   -0.000004
    4.154534    0.000906    0.000904    0.000002
    4.826900    0.000336    0.000336   -0.000000
    5.608078    0.000108    0.000110   -0.000002
    6.515677    0.000029    0.000029   -0.000000
    7.570157    0.000006    0.000006    0.000000
    8.795288    0.000001    0.000001    0.000000
    q(fit):     1.241803    rms diff:   0.000017
    fit: r>rmt  1.241803   r<rmt  3.486916   qtot  4.728719
    rho: r>rmt  1.241803   r<rmt  9.758197   qtot 11.000000
 tailsm: end

 coretail: q=0.00464, rho(rmt)=0.00535.  Fit with Hankel e=-24.113  coeff=770.4
      r            rhoc          fit
    2.280000    0.02411508    0.02411508
    2.457586    0.01084310    0.01086777
    2.649002    0.00457091    0.00457610
    2.855327    0.00179819    0.00179084
    3.077722    0.00065680    0.00064766
    3.317437    0.00022148    0.00021513
    3.575823    0.00006853    0.00006520
    3.854333    0.00001932    0.00001790
    4.154534    0.00000493    0.00000442
    4.478116    0.00000113    0.00000097
    4.826900    0.00000023    0.00000019
    5.202849    0.00000004    0.00000003
 end of freats: spid=Cu      

  Write mtopara.* ...

 Sum of reference energies: 0
 Exit 0 LMFA 
