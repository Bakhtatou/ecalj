#include "hip/hip_runtime.h"
 -----------------------  START LMFA (80000K)  -----------------------
 HEADER Cu

 rdctrl: reset global max nl from 5 to 6

 LMFA:     alat = 6.798  nbas = 1  nspec = 1  vn 7.00(LMFA 7.0)  verb 31,20
 pot:      XC:BH

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-08   alat= 6.79800   awald= 0.467
         r1=  1.959   nkd= 135      q1=  5.910   nkg= 181

 SGROUP: 1 symmetry operations from 0 generators
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 GROUPG: the following are sufficient to generate the space group:
         i*r3(1,1,-1) r4x
         i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 
conf:SPEC_ATOM= Cu : --- Table for atomic configuration ---
conf int(P)z = int(P) where P is replaced by PZ if it is semicore
conf:  isp  l  int(P) int(P)z    Qval    Qcore   CoreConf
conf:    1  0       4  4        1.000    6.000 => 1,2,3,
conf:    1  1       4  4        0.000   12.000 => 2,3,
conf:    1  2       3  3       10.000    0.000 => 
conf:    1  3       4  4        0.000    0.000 => 
conf:    1  4       5  5        0.000    0.000 => 
conf:    1  5       6  6        0.000    0.000 => 
conf:-----------------------------------------------------

 Species Cu:  Z=29  Qc=18  R=2.280000  Q=0
 mesh:   rmt=2.280000  rmax=48.145529  a=0.025  nr=393  nr(rmax)=515
  Pl=  4.5     4.5     3.5     4.5     5.5     6.5    
  Ql=  1.0     0.0     10.0    0.0     0.0     0.0    

  iter     qint         drho          vh0          rho0          vsum     beta
    1   29.000000   4.725E+03      145.0000    0.1442E+03      -58.2780   0.30
   51   29.000000   4.214E-05      274.8263    0.2633E+05     -130.7925   0.30


 sumev=-4.333255  etot=-3304.416258  eref=0.000000

 Free-atom wavefunctions:
 valence:      eval       node at      max at       c.t.p.   rho(r>rmt)
   4s      -0.36411         0.890       2.256       3.581     0.655341
   4p      -0.06295         0.975       3.484       7.413     0.906557
   3d      -0.39691         0.000       0.600       3.429     0.058644
   4f       0.02001         0.000      34.923      48.146*    1.000000
   5g       0.02754         0.000      36.368      48.146*    1.000000
   65       0.03607         0.000      37.464      48.146*    1.000000

 core:        ecore       node at      max at       c.t.p.   rho(r>rmt)
   1s    -649.07634         0.000       0.034       0.069     0.000000
   2s     -77.91382         0.070       0.197       0.308     0.000000
   2p     -67.32532         0.000       0.158       0.335     0.000000
   3s      -8.39248         0.288       0.614       0.895     0.000167
   3p      -5.29682         0.260       0.619       1.078     0.000836

 Optimise free-atom basis for species Cu, rmt=2.28
 l  it    Rsm      Eh     stiffR   stiffE      Eval      Exact     Pnu    Ql
 0   9   2.280  -0.284     108.7    530.9   -0.36392  -0.36411    4.75   1.00
 ... rsm exceeded rmt*2/3 .. repeat with rsm=rmt
 0   5   1.520  -0.129     108.7   4188.6   -0.35161  -0.36411    4.75   1.00
 1  11   2.280  -0.100     159.1      6.2   -0.04874  -0.06295    4.55   0.00
 ... rsm exceeded rmt*2/3 .. repeat with rsm=rmt
 1   1   1.520  -0.100     159.1  -2032.6    0.05991  -0.06295    4.55   0.00
 2  27   0.951  -0.112     175.3    114.6   -0.39668  -0.39691    3.89  10.00
 l=4  increase Pnu=   5.091  to    5.100
 l=5  increase Pnu=   6.069  to    6.100
 eigenvalue sum:  exact  -4.33326    opt basis  -4.31841    error 0.01484

 tailsm: fit tails to 6 smoothed hankels, rmt= 2.28000, rsm= 1.14000
 E:    -1.00000    -2.00000    -4.00000    -6.00000    -9.00000    -15.0000
 C:    -0.07287    10.70262    -181.908    1157.772    -4329.13    18357.95
        r          rho         fit         diff
    2.280000    0.018887    0.018854    0.000034
    2.927614    0.006048    0.006042    0.000005
    3.759167    0.001640    0.001641   -0.000001
    4.826901    0.000336    0.000336    0.000000
    6.197900    0.000046    0.000047   -0.000001
    7.958297    0.000004    0.000003    0.000000
    q(fit):     1.241803    rms diff:   0.000017
    fit: r>rmt  1.241803   r<rmt  3.487816   qtot  4.729619
    rho: r>rmt  1.241803   r<rmt  9.758197   qtot 11.000000

 coretail: q=0.00464, rho(rmt)=0.00535.  Fit with Hankel e=-24.113  coeff=770.|
      r            rhoc          fit
    2.280000    0.02411513    0.02411513
    2.396905    0.01425235    0.01427892
    2.716066    0.00337589    0.00337554
    3.077722    0.00065680    0.00064769
    3.487533    0.00010234    0.00009811
    3.951910    0.00001240    0.00001137
    4.478117    0.00000113    0.00000097
    5.074388    0.00000007    0.00000006
 
  Write mtopara.* ...

 Sum of reference energies: 0
 Exit 0 LMFA 
 wkinfo:  used    94 K  workspace of 80000 K   in   0 K calls
